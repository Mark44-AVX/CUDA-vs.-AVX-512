/**
 * This is a very basic sample that finds the coefficients of a regression line y = mx + b.
 * x and y are input vectors with numElements elements.
 */

#include <stdio.h>
#include <chrono>  // for high_resolution_clock struct and now()
using namespace std::chrono;

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel device code
 * vectorMult computes the element by element product of vectors A and B,
 *   storing the individual products in vector C.
 * Parameters
 *   C           - address of output vector of doubles.
 *   A           - address of first input vector of doubles.
 *   B           - address of second vector array of doubles.
 *   numElements - number of elements (doubles) in each vector.
 * No return value.
 * All 3 vectors are assumed to have the same number of elements.
 */
__global__ void
vectorMult(double *C, const double * A, const double * B, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		C[i] = A[i] * B[i];
	}
}


/**
* vectorSum computes the sum of the elements of a vector C.
* Parameters
*   C           - address of the vector of doubles.
*   numElements - number of elements in vector C.
* Returns the calculated sum of the elements of the passed vector.
*/
double vectorSum(const double *C, int numElements)
{
	double tempSum = 0.0;

	for (int i = 0; i < numElements; i += 8)
	{
		tempSum += C[i] + C[i + 1] + C[i + 2] + C[i + 3];
		tempSum += C[i + 4] + C[i + 5] + C[i + 6] + C[i + 7];
	}
	return tempSum;
}

/**
 * Host main routine
 */
int
main(void)
{
	// Local variables
	double h_sumX;  // Host-side sum of the x coordinates.
	double h_sumY;  // Host-side sum of the y coordinates.
	double h_sumXY; // Host-side sum of the xy pairs.
	double h_sumXX; // Host-side sum of the x^2 values.
	double m;       // Slope of the regression line.
	double b;       // y-intercept of the regression line.

	// Error code to check return values for CUDA calls.
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size.
	int numElements = 512 * 512;	// = 262144, or 2^18
	size_t size = numElements * sizeof(double);
	printf("[Linear regression of %d points]\n", numElements);

	// Allocate the host input vectors X, Y, XY, XX.
	// h_XY will eventually contain the term-by-term product of vectors X and Y.
	// h_XX will eventually contain the term-by-term product of vector X with itself.
	double *h_X = (double *)malloc(size);
	double *h_Y = (double *)malloc(size);
	double *h_XY = (double *)malloc(size);
	double *h_XX = (double *)malloc(size);


	// Verify that allocations succeeded.
	if (h_X == NULL || h_Y == NULL || h_XY == NULL || h_XX == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Declare and allocate the device input vector X.
	double *d_X = NULL;
	err = hipMalloc((void **)&d_X, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector X (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Declare and allocate the device input vector Y.
	double *d_Y = NULL;
	err = hipMalloc((void **)&d_Y, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector Y (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Declare and allocate the device output vector XY.
	double *d_XY = NULL;
	err = hipMalloc((void **)&d_XY, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector XY (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Declare and allocate the device output vector XX
	double *d_XX = NULL;
	err = hipMalloc((void **)&d_XX, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector XX (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors.
	// All points lie on the line y = 1.0*x + 0.5
	const double slope = 1.0;
	const double y_int = 0.5;

	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	
	// Fill the host-side vectors h_X and h_Y with data points.
	for (int i = 0; i < numElements; i += 4)
	{
		h_X[i] = slope * i;
		h_Y[i] = h_X[i] + y_int;
		h_X[i + 1] = slope * i;
		h_Y[i + 1] = h_X[i + 1] + y_int;
		h_X[i + 2] = slope * (i + 2);
		h_Y[i + 2] = h_X[i + 2] + y_int;
		h_X[i + 3] = slope * (i + 3);
		h_Y[i + 3] = h_X[i + 3] + y_int;
	}

	// Define a time point for a start time.
	high_resolution_clock::time_point t0 = high_resolution_clock::now();

	// Copy the host input vector X in host memory to the device input vector in
	// device memory.
	
	err = hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector X from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the host input vector Y in host memory to the device input vector in
	// device memory.
	//printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_Y, h_Y, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector Y from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	hipDeviceSynchronize();

	// Launch the vectorMult CUDA Kernel to calculate the vector XY.
	vectorMult <<<blocksPerGrid, threadsPerBlock >>> (d_XY, d_X, d_Y, numElements);
	hipDeviceSynchronize();

	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorMult kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}		

	hipDeviceSynchronize();
	
	// Launch the vectorMult CUDA Kernel to calculate the vector XX.
	vectorMult <<<blocksPerGrid, threadsPerBlock >>> (d_XX, d_X, d_X, numElements);
	hipDeviceSynchronize();	
	
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorMult kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the device vectors XY and XX to the corresponding host vectors.
	err = hipMemcpy(h_XY, d_XY, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector XY from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(h_XX, d_XX, size, hipMemcpyDeviceToHost);
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	auto et_vec_products = t1 - t0;
	auto et_vec_products_usec = et_vec_products / 1000;
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector XX from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Calculate the sums of the four vectors.
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	h_sumX = vectorSum(h_X, numElements);
	h_sumY = vectorSum(h_Y, numElements);
	h_sumXY = vectorSum(h_XY, numElements);
	h_sumXX = vectorSum(h_XX, numElements);	
	high_resolution_clock::time_point t3 = high_resolution_clock::now();
	auto et_vec_sums = t3 - t2;
	auto et_vec_sums_usec = et_vec_sums / 1000;

	printf("Sum of x: %f\n", h_sumX);
	printf("Sum of y: %f\n", h_sumY);
	printf("Sum of xy: %f\n", h_sumXY);
	printf("Sum of x^2: %f\n", h_sumXX);
	printf("Processed %d points\n", numElements);
	m = (numElements * h_sumXY - h_sumX * h_sumY) / (numElements * h_sumXX - h_sumX * h_sumX);
	b = (h_sumY - h_sumX) / numElements;

	// Display times.
	long long et_prods = et_vec_products.count();
	long long et_sums = et_vec_sums.count();

	printf("Time to calculate XY and XX: %lld nsec. = %lld usec.\n", et_prods, et_prods / 1000);
	printf("Time to calculate vector sums: %lld nsec. = %lld usec.\n", et_sums, et_sums / 1000);
	
	// Verify that the results are correct.
	printf("Predicted value of m: %lf\n", slope);
	printf("Computed value of m: %0.10lf\n", m);
	printf("Predicted value of b: %lf\n", y_int);
	printf("Computed value of b: %0.10lf\n", b);

	if (fabs(m - slope) > 1e-7 || fabs(b - y_int) > 1e-4)
	{
		fprintf(stderr, "Result verification failed!\n");
		exit(EXIT_FAILURE);
	}

	printf("Test PASSED\n");

	// Free device global memory for the vectors d_X, d_Y, d_XY, and d_XX.
	err = hipFree(d_X);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector X (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_Y);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector Y (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_XY);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector XY (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_XX);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector XX (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free host memory - vectors h_X, h_Y, h_XY, and h_XX.
	free(h_X);
	free(h_Y);
	free(h_XY);
	free(h_XX);

	// Reset the device and exit.
	// cudaDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling cudaDeviceReset causes all profile data to be
	// flushed before the application exits.
	err = hipDeviceReset();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Done\n");
}
